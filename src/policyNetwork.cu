#include <sjsuCuDNN/linearKernel.cuh>
#include <sjsuCuDNN/linear.h>
#include <sjsuCuDNN/reluKernel.cuh>
#include <sjsuCuDNN/relu.h>
#include <sjsuCuDNN/softmaxKernel.cuh>
#include <sjsuCuDNN/softmax.h>
#include <iostream>

PolicyNetwork::PolicyNetwork(int input_size, int hidden_size, int output_size) {
    this->input_size_ = input_size;
    this->hidden_size_ = hidden_size;
    this->output_size_ = output_size;

    // Initialize weights and biases
    hipMalloc(&weight1, hidden_size * input_size * sizeof(float));
    hipMalloc(&bias1, hidden_size * sizeof(float));
    hipMalloc(&weight2, output_size * hidden_size * sizeof(float));
    hipMalloc(&bias2, output_size * sizeof(float));

    // Initialize weights and biases with random values

    // Transfer weights and biases from host to device
}

virtual ~PolicyNetwork::PolicyNetwork() {
    hipFree(weight1_);
    hipFree(bias1_);
    hipFree(weight2_);
    hipFree(bias2_);
}

void PolicyNetwork::forward(const float* input, float* action_probs) {
    float* hidden = new float[hidden_size_];
    float* logits = new float[output_size_];

    // Perform Linear Transformation1; 1 for batch size
    linearLayer(input, weight1_, bias1_, hidden, 1, input_size_, hidden_size_);
    reluActivation(hidden, hidden, hidden_size_);

    // Perform Linear Transformation2
    linearLayer(hidden, weight2_, bias2_, logits, 1, hidden_size_, output_size_);
    softmaxActivation(logits, hidden, output_size_)

    delete[] hidden;
    delete[] logits;
}


void PolicyNetwork::linearLayer(const float* input, const float* weight, const float* bias,
                     float* output, int batch_size, int input_size int output_size) {
    linear(input, weight, bias, output, batch_size, input_size, output_size);
}

void PolicyNetwork::reluActivation(const float* input, float* output, int size) {
    relu(input, output, size);
}

void PolicyNetwork::softmaxActivation(const float* input, float* output, int size) {
    softmax(input, output, size);
}
