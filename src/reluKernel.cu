#include "hip/hip_runtime.h"
#include "sjsuCuDNN/reluKernel.cuh"

__global__ void reluKernel(const float* input, float* output, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < size) {
        output[index] = max(0.0f, input[index]);
    }
}
