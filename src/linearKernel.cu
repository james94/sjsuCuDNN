#include "hip/hip_runtime.h"
#include "sjsuCuDNN/linearKernel.cuh"

__global__ void linearKernel(const float* input, const float* weight, const float* bias,
                             float* output, int input_size int output_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < output_size) {
        float sum = 0.0f;
        for(int i = 0; i < input_size; i++) {
            sum += input[i] * weight[index * input_size + i];
        }
        output[index] = sum + bias[index];
    }
}