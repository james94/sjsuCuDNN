#include <sjsuCuDNN/softmaxKernel.cuh>
#include <sjsuCuDNN/softmax.h>
#include <iostream>

void softmax(const float* host_input, float* host_output, int size) {
    const int threads_per_block = 256;
    const int num_blocks = (size + threads_per_block - 1) / threads_per_block;

    // Allocate memory on the device for input and output
    float* device_input;
    float* device_output;
    hipMalloc(&device_input, size * sizeof(float));
    hipMalloc(&device_output, size * sizeof(float));

    // Transfer input from host to device
    hipMemcpy(device_input, host_input, size * sizeof(float), hipMemcpyHostToDevice);

    softmaxKernel<<<num_blocks, threads_per_block>>>(device_input, device_output, size);

    // Transfer the output from device to host
    hipMemcpy(host_output, device_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);
}
