#include <sjsuCuDNN/reluKernel.cuh>
#include <sjsuCuDNN/relu.h>
#include <iostream>

void relu(const float* host_input, float* host_output, int size) {
    const int threads_per_block = 256;
    const int num_blocks = (size + threads_per_block - 1) / threads_per_block;

    // Allocate memory on the device for input and output
    float* device_input;
    float* device_output;
    hipMalloc(&device_input, size * sizeof(float));
    hipMalloc(&device_output, size * sizeof(float));

    // Transfer input from host to device using hipMemcpy
    hipMemcpy(device_input, host_input, size * sizeof(float), hipMemcpyHostToDevice);
    // Ensure that all data transfers to device are done before proceeding
    hipDeviceSynchronize();
    // Launch the CUDA kernel
    reluKernel<<<num_blocks, threads_per_block>>>(device_input, device_output, size);
    // Transfer the output data from device to host using hipMemcpy
    hipMemcpy(host_output, device_output, size * sizeof(float), hipMemcpyDeviceToHost);
    // Ensure data transfer to host is done before proceeding
    hipDeviceSynchronize();

    // Free allocated memory on the device
    hipFree(device_input);
    hipFree(device_output);
}
