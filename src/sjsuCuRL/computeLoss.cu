#include "sjsuCuRL/computeLossKernel.cuh"
#include "sjsuCuRL/computeLoss.cuh"

void computeLoss(const float* states, const int* actions, const float* rewards, 
    float* action_probs, float* expected_return, float* loss, int num_samples, 
    PolicyNetwork* policy_net, int input_size, int output_size) {
    int block_size = 256;
    int grid_size = (num_samples + block_size - 1) / block_size;

    computeLossKernel<<<grid_size, block_size>>>(states, actions, rewards, action_probs,
        expected_return, loss, num_samples, policy_net, input_size, output_size);

    hipDeviceSynchronize();
}
