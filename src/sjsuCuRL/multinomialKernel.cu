#include "hip/hip_runtime.h"
#include "sjsuCuRL/multinomialKernel.cuh"

__global__ void multinomialKernel(const float* action_probs, int num_actions, int num_samples, int* actions) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < num_samples) {
        float rand_val = hiprand_uniform(&curand_state[index]);
        float cum_prob = 0.0;
        for(int action = 0; action < num_actions; ++action) {
            cum_prob += action_probs[action];
            if(rand_val < cum_prob) {
                actions[index] = action;
                break;
            }
        }
    }
}