#include "hip/hip_runtime.h"
#include "sjsuCuRL/computeExpectedReturn.cuh"
#include "sjsuCuRL/computeLossKernel.cuh"

__global__ void computeLossKernel(const float* states, const int* actions, const float* rewards, 
    float* action_probs, float* expected_return, float* loss, int num_samples, PolicyNetwork* policy_net,
    int input_size, int output_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < num_samples) {
        // Get current sample's data
        const float* state = states + tid * input_size;
        int action = actions[tid];
        float reward = rewards[tid];

        // Forward pass to compute the action probabilities
        network.forward(state, action_probs + tid * output_size);

        // Compute the negative log-likelihodd of the chosen action
        float neg_log_prob = -logf(action_probs[tid * output_size + action]);

        // Compute the expected return
        expected_return[tid] = computeExpectedReturn(rewards + tid, num_samples - tid);

        // Compute the loss
        loss[tid] = neg_log_prob * expected_return[tid]; 
    }
}
