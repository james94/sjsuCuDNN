#include "hip/hip_runtime.h"
#include "sjsuCuDNN/softmaxKernel.cuh"

__global__ void softmaxKernel(const float* input, float* output, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < size) {
        float max_val = input[index];
        for(int i = 0; i < size; i++) {
            max_val = max(max_val, input[i]);
        }

        float sum_exp = 0.0f;
        for(int i = 0; i < size; i++) {
            sum_exp += expf(input[i] - max_val);
        }

        output[index] = expf(input[index] - max_val) / sum_exp;
    }
}
