#include <sjsuCuDNN/linearKernel.cuh>
#include <sjsuCuDNN/linear.h>
#include <iostream>

void linear(const float* input, const float* weight, const float* bias,
            float* output, int batch_size, int input_size int output_size) {
    const int threads_per_block = 256;
    const int num_blocks = (output_size + threads_per_block - 1) / threads_per_block;

    // Allocate memory on the device for input, weight, bias and output
    float* device_input;
    float* device_weight;
    float* device_bias;
    float* device_output;
    hipMalloc(&device_input, batch_size * input_size * sizeof(float));
    hipMalloc(&device_weight, input_size * output_size * sizeof(float));
    hipMalloc(&device_bias, output_size * sizeof(float));
    hipMalloc(&device_output, batch_size * output_size * sizeof(float));

    // Transfer input, weight and bias from host to device using hipMemcpy
    hipMemcpy(device_input, input, batch_size * input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_weight, weight, input_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_bias, bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    linearKernel<<<num_blocks, threads_per_block>>>(device_input, device_weight, device_bias,
                                                    device_output, input_size, output_size);

    // Transfer the output from device to host 
    hipMemcpy(output, device_output, batch_size * output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free allocated memory on the device
    hipFree(device_input);
    hipFree(device_weight);
    hipFree(device_bias);
    hipFree(device_output);
}
