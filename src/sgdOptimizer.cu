#include <sjsuCuDNN/sgdUpdateKernel.cuh>
#include <sjsuCuDNN/sgdOptimizer.cuh>

void SGDOptimizer::step() {
    sgdUpdate();
}

/*
    PolicyNetwork& policy_net_;
    float learning_rate_;
*/
void SGDOptimizer::sgdUpdate() {
    const std::vector<float>& gradients = policy_net_.getGradients(); // no method yet

    // Allocate device memory for parameters and gradients
    float* d_parameters;
    float* d_gradients;
    hipMalloc(&d_parameters, gradients.size() * sizeof(float));
    hipMalloc(&d_gradients, gradients.size() * sizeof(float));

    // Transfer data from host to device
    // no getParameters() method yet
    hipMemcpy(d_paramters, policy_net_.getParameters(), gradients.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gradients, gradients.data(), gradients.size() * sizeof(float), hipMemcpyHostToDevice);

    // Launch the SGD update kernel
    int blockSize = 256;
    int numBlocks = (gradients.size() + blockSize - 1) / blockSize;
    sgdUpdateKernel<<<numBlocks, blockSize>>>(d_parameters, d_gradients, learning_rate_, gradients.size());

    // Transfer updated parameters from device back to host
    hipMemcpy(policy_net_.getParameters(), d_parameters, gradients.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_parameters);
    hipFree(d_gradients);
}
