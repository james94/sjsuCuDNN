#include "hip/hip_runtime.h"
#include "sjsuCuDNN/sgdUpdateKernel.cuh"

__global__ void sgdUpdateKernel(float* parameters, const float* gradients, float learning_rate, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < size) {
        parameters[index] -= learning_rate * gradients[index];
    }
}
