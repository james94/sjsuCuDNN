#include <gtest/gtest.h>
#include "sjsuCuRL/multinomialKernel.cuh"
#include "sjsuCuRL/policyNetwork.cuh"
#include <opencv2/opencv.hpp>

TEST(MultinomialSamplingTest, SampledActionsValidation) {
    // Define the hyperparameters
    int input_size = 4;
    int hidden_size = 64;
    int output_size = 6;
    int num_samples = 100;

    PolicyNetwork policy_net(input_size, hidden_size, output_size);

    // Generate example input data
    cv::Mat image = cv::Mat::zeros(input_size, 1, CV_32FC1);
    float* state = image.ptr<float>();

    // Allocate device memory for action porobabilities and actions
    float* d_state;
    float* d_action_probs;

    hipMalloc(&d_state, input_size * sizeof(float));
    hipMalloc(&d_action_probs, output_size * sizeof(float));

    // Transfer data from host to device
    hipMemcpy(d_state, state, input_size * sizeof(float), hipMemcpyHostToDevice);

    policy_net.forward(d_state, d_action_probs);

    // Allocate device memory for sampled actions
    int* d_actions;
    hipMalloc(&d_actions, num_samples * sizeof(int));


    // Launch the multinomial sampling
    int blockSize = 256;
    int numBlocks = (num_samples + blockSize - 1) / blockSize;
    multinomialKernel<<<numBlocks, blockSize>>>(d_action_probs, output_size, num_samples, d_actions);

    // Transfer data from device to host
    int* sampled_actions = new int[num_samples];
    hipMemcpy(sampled_actions, d_actions, num_samples * sizeof(int), hipMemcpyDeviceToHost);

    // Validate the sampled actions
    for(int i = 0; i < num_samples; ++i) {
        ASSERT_GE(sampled_actions[i], 0);
        ASSERT_LT(sampled_actions[i], output_size);
    }

    // Free device memory
    hipFree(d_state);
    hipFree(d_action_probs);
    hipFree(d_actions); 
    delete[] sampled_actions;
}