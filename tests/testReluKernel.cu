#include <gtest/gtest.h>
#include "sjsuCuDNN/reluKernel.cuh"

TEST(ReluKernelTest, OutputValidation) {
    const int size = 10;
    const int block_size = 256;

    float* input;
    float* output;
    
    // Allocate memory on the device
    hipMalloc(&input, size * sizeof(float));
    hipMalloc(&output, size * sizeof(float));

    // Initialize input data on the host
    float host_input[size] = {-1.0, 2.0, -3.0, 4.0, -5.0, 6.0, -7.0, 8.0, -9.0, 10.0};

    // Copy input data from host to device
    hipMemcpy(input, host_input, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    reluKernel<<<(size + block_size - 1) / block_size, block_size>>>(input, output, size);

    // Copy output data from device to host
    float host_output[size];
    hipMemcpy(host_output, output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Validate the output
    float expected_output[size] = {0.0, 2.0, 0.0, 4.0, 0.0, 6.0, 0.0, 8.0, 0.0, 10.0};
    for(int i = 0; i < size; ++i) {
        EXPECT_FLOAT_EQ(host_output[i], expected_output[i]);
    }

    // Free allocated memory on the device
    hipFree(input);
    hipFree(output);
}
